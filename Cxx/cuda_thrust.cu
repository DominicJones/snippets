#include "hip/hip_runtime.h"
// -*- C++ -*-

// nvcc -std=c++14 -I$CUDART_PATH/include -L$CUDART_PATH/lib64 <file>.cu

#include "hip/hip_runtime.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <iostream>
#include <chrono>
#include <vector>


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
  {
    y[i] = a*x[i] + y[i];
  }
}


int main()
{
  using Clock = std::chrono::high_resolution_clock;

  int N = 10000000;
  int n_threads_per_block = 256;
  int n_blocks = (N + n_threads_per_block - 1) / n_threads_per_block;

  thrust::host_vector<float> hx, hy;
  hx.resize(N);
  hy.resize(N);

  for (int i = 0; i < N; i++) {
    hx[i] = 1;
    hy[i] = 2;
  }

  thrust::device_vector<float> dx, dy;
  dx = hx;
  dy = hy;

  auto timer_start = Clock::now();

  auto dxp = thrust::raw_pointer_cast(&dx[0]);
  auto dyp = thrust::raw_pointer_cast(&dy[0]);
  saxpy<<<n_blocks, n_threads_per_block>>>(N, 2, dxp, dyp);

  auto timer_end = Clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(timer_end - timer_start).count();

  thrust::copy(dy.begin(), dy.end(), hy.begin());

  float maxError = 0;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(hy[i] - 4));

  std::cout << "max error: " << maxError << std::endl;
  std::cout << "timer: "  << duration << " ns" << std::endl;
}
